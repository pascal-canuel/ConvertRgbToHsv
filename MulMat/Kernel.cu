#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "hip/hip_runtime.h"
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/video/tracking.hpp>
#include "stdafx.h"
//#include "nppdefs.h"
//#include <npp.h>

typedef unsigned char uchar;
typedef unsigned int uint;

#define BLOCK_SIZE 32
#define CV_64FC1 double
#define CV_32F float
#define CV_8U uchar

int iDivUp(int a, int b)
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__device__
double maxVal(double blue, double green, double red) {
	if ((blue >= green) && (blue >= red))
		return blue;
	else if ((green >= blue) && (green >= red))
		return green;
	else
		return red;
}

__device__
double minVal(double blue, double green, double red) {
	if ((blue <= green) && (blue <= red))
		return blue;
	else if ((green <= blue) && (green <= red))
		return green;
	else
		return red;
}

// Transfert img to imgout to see how opencv image can be acces in GPGPU
__global__ void Kernel_Tst_Img_CV_8U(uchar *img, uchar *imgout, int ImgWidth, int imgHeigh)
{
	int ImgNumColonne = blockIdx.x  * blockDim.x + threadIdx.x;
	int ImgNumLigne = blockIdx.y  * blockDim.y + threadIdx.y;
	int Index = (ImgNumLigne * ImgWidth + ImgNumColonne * 3);

	if ((ImgNumColonne < ImgWidth / 3) && (ImgNumLigne < imgHeigh))
	{
		/* Kernel Code Here */

		double blue = (double)img[Index] / 255;
		double green = (double)img[Index + 1] / 255;
		double red = (double)img[Index + 2] / 255;

		double cMax = maxVal(blue, green, red);

		double cMin = minVal(blue, green, red);

		double delta = cMax - cMin;

		//	HUE
		double hue = 0;
		if (blue == cMax) {
			hue = 60 * ((red - green) / delta + 4);
		}
		else if (green == cMax) {
			hue = 60 * ((blue - red) / delta + 2);
		}
		else if (red == cMax) {
			hue = 60 * ((green - blue) / delta);
			if (hue < 0)
				hue += 360;
		}

		//	SATURATION
		double saturation = 0;
		if (cMax != 0) {
			saturation = delta / cMax;
		}

		//	VALUE
		double value = cMax;

		imgout[Index] = (uchar)(hue / 2);
		imgout[Index + 1] = (uchar)(saturation * 255);
		imgout[Index + 2] = (uchar)(value * 255);
	}

	return;
}

extern "C" bool GPGPU_TstImg_CV_8U(cv::Mat* img, cv::Mat* GPGPUimg)
{
	hipError_t cudaStatus;
	uchar *devImage;
	uchar *devImageOut;

	unsigned int ImageSize = img->rows * img->step1();// step number of bytes in each row

													  // Allocate memory for image
	cudaStatus = hipMalloc((void**)&devImage, ImageSize);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Upload the image to the GPU
	cudaStatus = hipMemcpy(devImage, img->data, ImageSize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	//dim3 dimGrid(iDivUp(img->step1(), BLOCK_SIZE), iDivUp(img->cols, BLOCK_SIZE));
	dim3 dimGrid(iDivUp(img->cols, BLOCK_SIZE), iDivUp(img->rows, BLOCK_SIZE));


	// Test only
	// Allocate memory for the result image 
	cudaStatus = hipMalloc((void**)&devImageOut, ImageSize);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	Kernel_Tst_Img_CV_8U << <dimGrid, dimBlock >> >(devImage, devImageOut, img->step1(), img->rows);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//Wait for the kernel to end
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize failed!");
		goto Error;
	}

	// Download the result image from gpu
	cudaStatus = hipMemcpy(GPGPUimg->data, devImageOut, ImageSize, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	return true;
Error:
	hipFree(devImage);
	hipFree(devImageOut);

	return cudaStatus;
}
// Transfert img to imgout to see how opencv image can be acces in GPGPU

__global__ void Kernel_Sobel()
{

}

//	TODO add color
__global__ void Kernel_ThresholdHSV(uchar *img, uchar *imgout, int ImgWidth, int imgHeigh, int minHue, int maxHue)
{
	int ImgNumColonne = blockIdx.x  * blockDim.x + threadIdx.x;
	int ImgNumLigne = blockIdx.y  * blockDim.y + threadIdx.y;
	int Index = (ImgNumLigne * ImgWidth) + (ImgNumColonne * 3);

	if ((ImgNumColonne < ImgWidth / 3) && (ImgNumLigne < imgHeigh))
	{
		int hue = img[Index];
		int saturation = img[Index + 1];
		int value = img[Index + 2];

		if (hue > minHue && hue < maxHue) {
			imgout[Index] = 0;
			imgout[Index + 1] = 0;
			imgout[Index + 2] = 0;
		}
		else {
			imgout[Index] = 255;
			imgout[Index + 1] = 255;
			imgout[Index + 2] = 255;
		}
	}

	return;
}

extern "C" bool GPGPU_BackGroundSubstractionHSV(cv::Mat* imgHSV, cv::Mat* GPGPUimg, int minHue, int maxHue,
	cv::Scalar backGroundColor, bool replaceForeground = false, cv::Scalar ForegroundColor = cv::Scalar(0, 0, 0))
{
	//	1. Initialize data
	hipError_t cudaStatus;
	uchar *gDevImage;
	uchar *gDevImageOut;

	uint imageSize = imgHSV->rows * imgHSV->step1();

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(iDivUp(imgHSV->cols, BLOCK_SIZE), iDivUp(imgHSV->rows, BLOCK_SIZE));

	//	2. Allocation data
	cudaStatus = hipMalloc(&gDevImage, imageSize);
	cudaStatus = hipMalloc(&gDevImageOut, imageSize);

	//	3. Copy data on GPU
	cudaStatus = hipMemcpy(gDevImage, imgHSV->data, imageSize, hipMemcpyHostToDevice);

	//	4. Launch kernel
	Kernel_ThresholdHSV << <dimGrid, dimBlock >> >(gDevImage, gDevImageOut, imgHSV->step1(), imgHSV->rows, 38, 95);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//Wait for the kernel to end
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize failed!");
		goto Error;
	}

	//	5. Copy data on CPU
	cudaStatus = hipMemcpy(GPGPUimg->data, gDevImageOut, imageSize, hipMemcpyDeviceToHost);

	//	6. Free GPU memory
	hipFree(gDevImage);
	hipFree(gDevImageOut);

	return true;
Error:
	hipFree(gDevImage);
	hipFree(gDevImageOut);

	return cudaStatus;
}
